
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
// #include "helpers.cuh"
using namespace std;

#define NUMPAD 128

typedef struct _Node_info{
    u_short parent_index;
    u_int potential_flow;
} Node_info;


__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks);
__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited, u_int total_nodes, u_int sink, u_int* locks);
__global__ void augment_path(Node_info* node_infos, bool* do_change_capacity , u_int total_nodes, u_short* residual_capacity, u_int bottleneck_flow);
void initialization_device(u_short* d_residual_capacity, u_short* residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* d_visited, bool* d_do_change_capacity, u_int* d_locks, size_t* matrix_size, size_t* node_infos_size, size_t* vertices_size, size_t* locks_size);
void reset_device(Node_info* d_node_info, bool* d_frontier, bool* d_visited, int source, int N, u_int* d_locks);
void find_augmenting_path_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* frontier, bool* d_visited, size_t* vertices_size, u_int N, u_int sink, u_int* d_locks);
void D2H_node_info(Node_info* node_info, Node_info* d_node_info, size_t* node_infos_size);
void augment_path_device(bool* do_change_capacity, Node_info* d_node_info, bool* d_do_change_capacity, u_int N, u_short* d_residual_capacity, u_int bottleneck_flow, size_t* vertices_size);
void free_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* d_visited);


__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < total_nodes){
        frontier[id] = (id == source);
        visited[id] = false;
        node_info[id].potential_flow = UINT_MAX;
        locks[id] = 0;
    }
}

__global__ void augment_path(Node_info* node_infos, bool* do_change_capacity , u_int total_nodes, u_short* residual_capacity, u_int bottleneck_flow) {
    int node_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(node_id < total_nodes && do_change_capacity[node_id]){
        Node_info* current_node_info = node_infos + node_id;
        residual_capacity[current_node_info->parent_index * total_nodes + node_id] -= bottleneck_flow;
        residual_capacity[node_id * total_nodes + current_node_info->parent_index] += bottleneck_flow;
    }
}

__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited, u_int total_nodes, u_int sink, u_int* locks) {

    int node_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(!frontier[sink] && node_id < total_nodes && frontier[node_id]){

        frontier[node_id] = false;
        visited[node_id] = true;

        Node_info *neighbour;
        Node_info current_node_info = node_info[node_id];
        u_int capacity;

        for (u_int i = node_id; i < total_nodes; ++i){

            if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                continue;
            }

            frontier[i] = true;
            locks[i] = 0;

            neighbour = node_info + i;
            neighbour->parent_index = node_id;
            neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
        }


        for (u_int i = 0; i < node_id; ++i){

            if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                continue;
            }

            frontier[i] = true;
            locks[i] = 0;

            neighbour = node_info + i;
            neighbour->parent_index = node_id;
            neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
        }
    }
}

void initialization_device(
    u_short* d_residual_capacity,
    u_short* residual_capacity,
    Node_info* d_node_info,
    bool* d_frontier,
    bool* d_visited,
    bool* d_do_change_capacity,
    u_int* d_locks,
    size_t* matrix_size,
    size_t* node_infos_size,
    size_t* vertices_size,
    size_t* locks_size
) {
    hipMalloc((void **)&d_residual_capacity, *matrix_size);
    hipMalloc((void **)&d_node_info, *node_infos_size);
    hipMalloc((void **)&d_frontier, *vertices_size);
    hipMalloc((void **)&d_visited, *vertices_size);
    hipMalloc((void **)&d_do_change_capacity, *vertices_size);
    hipMalloc((void **)&d_locks, *locks_size);

    hipMemcpy(d_residual_capacity, residual_capacity, *matrix_size, hipMemcpyHostToDevice);
}

void reset_device(Node_info* d_node_info, bool* d_frontier, bool* d_visited, int source, int N, u_int* d_locks) {
    int num_threads = 1024;
    int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    dim3 threads(num_threads);
    dim3 blocks(num_blocks);

    reset<<<blocks, threads>>>(d_node_info, d_frontier, d_visited, source, N, d_locks);
}

void find_augmenting_path_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* frontier, bool* d_visited, size_t* vertices_size, u_int N, u_int sink, u_int* d_locks) {
    int num_threads = 1024;
    int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    dim3 threads(num_threads);
    dim3 blocks(num_blocks);

    // Invoke kernel
    find_augmenting_path<<<blocks, threads>>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks);

    // Copy back frontier from device
    hipMemcpy(frontier, d_frontier, *vertices_size, hipMemcpyDeviceToHost);
}

void D2H_node_info(Node_info* node_info, Node_info* d_node_info, size_t* node_infos_size) {
    // copy node_info from device to host
    hipMemcpy(node_info, d_node_info, *node_infos_size, hipMemcpyDeviceToHost);
}

void augment_path_device(bool* do_change_capacity, Node_info* d_node_info, bool* d_do_change_capacity, u_int N, u_short* d_residual_capacity, u_int bottleneck_flow, size_t* vertices_size) {
    int num_threads = 1024;
    int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    dim3 threads(num_threads);
    dim3 blocks(num_blocks);

    hipMemcpy(d_do_change_capacity, do_change_capacity, *vertices_size, hipMemcpyHostToDevice);
    augment_path<<< blocks, threads >>>(d_node_info, d_do_change_capacity, N, d_residual_capacity, bottleneck_flow);
}

void free_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* d_visited) {
    hipFree(d_residual_capacity);
    hipFree(d_node_info);
    hipFree(d_frontier);
    hipFree(d_visited);
}