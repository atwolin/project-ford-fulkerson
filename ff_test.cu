#include "hip/hip_runtime.h"
#include <bit/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include "helpers.cuh"
using namespace std;

#define milliseconds 1e3
#define NUMPAD 128



u_int N;

typedef struct _Node_info {
    u_short parent_index;
    u_int potential_flow;
} Node_info;
void input(const char* filename, u_int total_nodes, u_short* residual_capacity);
void output(char* filename, int max_flow, double time);

__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks);

int main(int argc, char** argv) {
    if (argc < 4) {
        printf("Required input filename, data size, and output filename")
    }
    Timer timer;

    N = atoi(argv[2]);
    // u_int V = atoi(argv[2]);
    // N = (V % NUMPAD == 0) ? V : (V / NUMPAD + 1) * NUMPAD;
    size_t matrix_size = N * N sizeof(u_short);

    u_short *residual_capacity;
    residual_capacity = (u_short *)malloc(matrix_size);
    memset(residual_capacity, 0, matrix_size);

    input(argv[1], N, residual_capacity);

    u_int source = 0, sink = N - 1;
    u_int current_vertex, bottleneck_flow;
    u_int max_flow = 0;
    bool found_augmenting_path;

    size_t node_infos_size = N * sizeof(Node_info);
    size_t vertices_size = N * sizeof(bool);
    size_t locks_size = N * sizeof(u_int);


    Node_info* current_node_info;
    Node_info* node_info = (Node_info *)malloc(node_infos_size);
    bool* frontier = (bool *)malloc(vertices_size);
    bool* do_change_capacity = (bool *)malloc(vertices_size);

    u_short* d_residual_capacity;
    Node_info* d_node_info;
    bool* d_frontier, *d_visited, *d_do_change_capacity;
    u_int* d_locks;

    hipMalloc((void **)&d_residual_capacity, matrix_size);
    hipMalloc((void **)&d_node_info,node_infos_size);
    hipMalloc((void **)&d_frontier, vertices_size);
    hipMalloc((void **)&d_visited, vertices_size);
    hipMalloc((void **)&d_do_change_capacity, vertices_size);
    hipMalloc((void **)&d_locks, locks_size);
    printf("d_residual_capacity_size: %d,\nd_locks_size: %d,\nd_node_info_size: %d,\nd_frontier: %d,\nd_visited: %d,\nd_do_change_capacity: %d\n",
            matrix_size, locks_size, node_infos_size, vertices_size, vertices_size, vertices_size);

    hipMemcpy(d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);

    int num_threads = 1024;
    int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    dim3 threads(num_threads);
    dim3 blocks(num_blocks);

    do {
        reset<<<num_blocks, num_threads>>>(d_node_info, d_frontier, d_visited, source, N, d_locks, d_do_change_capacity);
    } while(found_augmenting_path);

}


void input(const char* filename, u_int total_nodes, u_short* residual_capacity) {
    ifstream file;
    file.open(filename);

    if (!file) {
        printf("Error reading file.\n");
        exit(true);
    }

    string line;
    u_int source, destination;
    u_short capacity;

    while (file) {
        getline(file, line);
        if (line.empty()) {
            continue;
        }
        stringstream linestream(line);
        linestream >> source >> destination >> capacity;
        residual_capacity[source * total_nodes + destination] = capacity;
    }
    file.close();
}

void output(char* filename, int max_flow, double time) {
    FILE* outfile = fopen(filename, "w");
    fprintf(outfile, "Max Flow: %d\n", max_flow);
    fprintf(outfile, "Time(ms): %.2f\n", time);
    fclose(outfile);
}

__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks, bool* do_change_capacity) {
    int id = threadIdx.x + blockIdx.x * blockDim.x;
    if(id < total_nodes){
        // frontier[id] = id == source;
        visited[id] = false;
        node_info[id].potential_flow = UINT_MAX;
        locks[id] = 0;
    }

    // in reset_host
    for (int i = 0; i < total_nodes; ++i) {
        frontier[i] = false;
        do_change_capacity[i] = false;
    }
}