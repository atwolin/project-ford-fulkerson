
#include <hip/hip_runtime.h>
#include <bits/stdc++.h>
#define milliseconds 1e3

using namespace std;

typedef struct _Node_info{
    u_short parent_index;
    u_int potential_flow;
} Node_info;

int num_threads = 1024;
// int num_blocks = 1;
dim3 threads(num_threads);
// dim3 blocks(num_blocks);

void readInput(const char* filename, u_int total_nodes, u_short* residual_capacity);
void output(char* outFileName, int max_flow, double time);

__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks);
__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited,
    u_int total_nodes, u_int sink, u_int* locks);
__global__ void augment_path(Node_info* node_infos, bool* do_change_capacity , u_int total_nodes, u_short* residual_capacity, u_int bottleneck_flow);

void initialization_device(u_short* residual_capacity,
                           u_short** d_residual_capacity, Node_info** d_node_info,
                           bool** d_frontier, bool** d_visited, bool** d_do_change_capacity,
                           u_int** d_locks,
                           size_t matrix_size, size_t node_infos_size, size_t vertices_size, size_t locks_size);

void reset_device(Node_info* d_node_info, bool* d_frontier, bool* d_visited, int source, int N, u_int* d_locks);
void reset_host(bool* frontier, int source, int total_nodes, bool* do_change_capacity);
bool is_frontier_empty_or_sink_found(bool* frontier, int N, int sink_pos);
void find_augmenting_path_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* d_visited,
                                 u_int N, u_int sink, u_int* d_locks,
                                 bool* frontier, size_t vertices_size);
void D2H_node_info(Node_info* node_info, Node_info* d_node_info, size_t node_infos_size);
void augment_path_device(bool* do_change_capacity, Node_info* d_node_info, bool* d_do_change_capacity, u_int N, u_short* d_residual_capacity, u_int bottleneck_flow, size_t vertices_size);
// void free_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* d_visited);
void free_device(u_short** d_residual_capacity, Node_info** d_node_info, bool** d_frontier, bool** d_visited);

int main(int argc, char** argv){

    if(argc < 4){
        printf("Specify filename & number of vertices\n");
        return 1;
    }

    u_int N = atoi(argv[2]);
    u_short *residual_capacity;

    size_t matrix_size = N * N * sizeof(u_short);
    residual_capacity = (u_short *)malloc(matrix_size);
    memset(residual_capacity, 0, matrix_size);

    readInput(argv[1], N, residual_capacity);

    u_int source=0, sink=N-1;
    u_int current_vertex, bottleneck_flow;
    u_int max_flow = 0;

    Node_info* current_node_info;
    u_short* d_residual_capacity;
    u_int* d_locks;
    bool* frontier;
    bool* d_frontier, *d_visited, *d_do_change_capacity, *do_change_capacity;

    Node_info* node_info;
    Node_info* d_node_info;

    clock_t start_time = clock();

    size_t node_infos_size = N * sizeof(Node_info);
    node_info = (Node_info*)malloc(node_infos_size);

    size_t vertices_size = N * sizeof(bool);
    frontier = (bool *)malloc(vertices_size);
    do_change_capacity = (bool *)malloc(vertices_size);

    size_t locks_size = N * sizeof(u_int);

    // cudaMalloc((void **)&d_residual_capacity, matrix_size);
    // cudaMalloc((void **)&d_locks, locks_size);
    // cudaMalloc((void **)&d_node_info,node_infos_size);
    // cudaMalloc((void **)&d_frontier, vertices_size);
    // cudaMalloc((void **)&d_visited, vertices_size);
    // cudaMalloc((void **)&d_do_change_capacity, vertices_size);

    // cudaMemcpy(d_residual_capacity, residual_capacity, matrix_size, cudaMemcpyHostToDevice);
	initialization_device(residual_capacity, &d_residual_capacity, &d_node_info,
                     &d_frontier, &d_visited, &d_do_change_capacity,
                     &d_locks,
					 matrix_size, node_infos_size, vertices_size, locks_size);

    bool found_augmenting_path;

    // int threads = 256;
    // int blocks = ceil(N * 1.0 /threads);
    // int num_threads = 1024;
    // int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    // dim3 threads(num_threads);
    // dim3 blocks(num_blocks);
    // num_threads = 1024;
    int num_blocks = (N / num_threads <= 0) ? 1 : (N + (num_threads - 1)) / num_threads;
    // threads(num_threads);
    dim3 blocks(num_blocks);


    do{
        // reset visited, frontier, node_info, locks
        // reset<<<blocks, threads >>>(d_node_info, d_frontier, d_visited, source, N, d_locks);
        reset_device(d_node_info, d_frontier, d_visited, source, N, d_locks);
        reset_host(frontier, source, N, do_change_capacity);

        while(!is_frontier_empty_or_sink_found(frontier, N, sink)){
                // // Invoke kernel
                // find_augmenting_path<<< blocks, threads >>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks);

                // // Copy back frontier from device
                // cudaMemcpy(frontier, d_frontier, vertices_size, cudaMemcpyDeviceToHost);
				find_augmenting_path_device(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks,
                          					frontier, vertices_size);
        }

        found_augmenting_path = frontier[sink];
        if(!found_augmenting_path){
            break;
        }

        // copy node_info from device to host
        // cudaMemcpy(node_info, d_node_info, node_infos_size, cudaMemcpyDeviceToHost);
		D2H_node_info(node_info, d_node_info, node_infos_size);

        bottleneck_flow = node_info[sink].potential_flow;
        max_flow += bottleneck_flow;

        for(current_vertex = sink; current_vertex != source; current_vertex = current_node_info->parent_index){
            current_node_info = node_info + current_vertex;
            do_change_capacity[current_vertex] = true;
        }

        // cudaMemcpy(d_do_change_capacity, do_change_capacity, vertices_size, cudaMemcpyHostToDevice);

        // augment_path<<< blocks, threads >>>(d_node_info, d_do_change_capacity, N, d_residual_capacity, bottleneck_flow);
		augment_path_device(do_change_capacity, d_node_info, d_do_change_capacity, N, d_residual_capacity, bottleneck_flow, vertices_size);

    } while(found_augmenting_path);

    printf("\nmaxflow %d\n", max_flow);
    double time_taken = ((double)clock() - start_time)/CLOCKS_PER_SEC * milliseconds; // in milliseconds
    printf("%f ms for thread size- %d\n", time_taken, num_threads);
    output(argv[3], max_flow, time_taken);


    free(residual_capacity);
    free(frontier);
    free(node_info);

    // cudaFree(d_residual_capacity);
    // cudaFree(d_node_info);
    // cudaFree(d_frontier);
    // cudaFree(d_visited);
	free_device(&d_residual_capacity, &d_node_info, &d_frontier, &d_visited);

    return 0;
}

void readInput(const char* filename, u_int total_nodes, u_short* residual_capacity) {

    ifstream file;
    file.open(filename);

    if (!file) {
        cout <<  "Error reading file!";
        exit(1);
    }

    string line;
    u_int source, destination;
    u_short capacity;

    while (file) {

        getline(file, line);

        if (line.empty()) {
            continue;
        }

        std::stringstream linestream(line);
        linestream >> source >> destination >> capacity;
        residual_capacity[source * total_nodes + destination] = capacity;
    }
    file.close();
}

void output(char* outFileName, int max_flow, double time) {
    FILE* outfile = fopen(outFileName, "w");
    fprintf(outfile, "Max Flow: %d\n", max_flow);
    fprintf(outfile, "Time: %.2f ms\n", time);
    fclose(outfile);
}

__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited,
    u_int total_nodes, u_int sink, u_int* locks) {

    int node_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(!frontier[sink] && node_id < total_nodes && frontier[node_id]){

        frontier[node_id] = false;
        visited[node_id] = true;

        Node_info *neighbour;
        Node_info current_node_info = node_info[node_id];
        u_int capacity;

        for (u_int i = node_id; i < total_nodes; ++i){

            if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                continue;
            }

            frontier[i] = true;
            locks[i] = 0;

            neighbour = node_info + i;
            neighbour->parent_index = node_id;
            neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
        }


        for (u_int i = 0; i < node_id; ++i){

            if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                continue;
            }

            frontier[i] = true;
            locks[i] = 0;

            neighbour = node_info + i;
            neighbour->parent_index = node_id;
            neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
        }
    }
}

__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks) {
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < total_nodes){
        frontier[id] = id == source;
        visited[id] = false;
        node_info[id].potential_flow = UINT_MAX;
        locks[id] = 0;

    }
}
__global__ void augment_path(Node_info* node_infos, bool* do_change_capacity , u_int total_nodes, u_short* residual_capacity, u_int bottleneck_flow) {
    int node_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(node_id < total_nodes && do_change_capacity[node_id]){
        Node_info* current_node_info = node_infos + node_id;
        residual_capacity[current_node_info->parent_index * total_nodes + node_id] -= bottleneck_flow;
        residual_capacity[node_id * total_nodes + current_node_info->parent_index] += bottleneck_flow;
    }
}

void initialization_device(u_short* residual_capacity,
                           u_short** d_residual_capacity, Node_info** d_node_info,
                           bool** d_frontier, bool** d_visited, bool** d_do_change_capacity,
                           u_int** d_locks,
                           size_t matrix_size, size_t node_infos_size, size_t vertices_size, size_t locks_size) {
	hipMalloc((void **)d_residual_capacity, matrix_size);
    hipMalloc((void **)d_node_info,node_infos_size);
    hipMalloc((void **)d_frontier, vertices_size);
    hipMalloc((void **)d_visited, vertices_size);
    hipMalloc((void **)d_do_change_capacity, vertices_size);
	hipMalloc((void **)d_locks, locks_size);

    hipMemcpy(*d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);
}

void reset_device(Node_info* d_node_info, bool* d_frontier, bool* d_visited, int source, int N, u_int* d_locks) {
    // num_threads = 1024;
    int num_blocks = (N / num_threads <= 0) ? 1 : (N + (num_threads - 1)) / num_threads;
    // threads(num_threads);
    dim3 blocks(num_blocks);

    reset<<<blocks, threads >>>(d_node_info, d_frontier, d_visited, source, N, d_locks);
}

void reset_host(bool* frontier, int source, int total_nodes, bool* do_change_capacity) {
    frontier[source] = true;
    do_change_capacity[source] = false;

    for (int i = source+1; i < total_nodes; i++) {
        frontier[i] = false;
        do_change_capacity[i] = false;
    }

    for (int i = 0; i < source; i++) {
        frontier[i] = false;
        do_change_capacity[i] = false;
    }
}

bool is_frontier_empty_or_sink_found(bool* frontier, int N, int sink_pos) {
    for (int i = N-1; i > -1; --i) {
        if(frontier[i]){
            return i == sink_pos;
        }
    }
    return true;
}

void find_augmenting_path_device(u_short* d_residual_capacity, Node_info* d_node_info, bool* d_frontier, bool* d_visited,
                          u_int N, u_int sink, u_int* d_locks,
                          bool* frontier, size_t vertices_size) {
	// num_threads = 1024;
    int num_blocks = (N / num_threads <= 0) ? 1 : (N + (num_threads - 1)) / num_threads;
    // threads(num_threads);
    dim3 blocks(num_blocks);

    // Invoke kernel
    find_augmenting_path<<< blocks, threads >>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks);

    // Copy back frontier from device
    hipMemcpy(frontier, d_frontier, vertices_size, hipMemcpyDeviceToHost);

}

void D2H_node_info(Node_info* node_info, Node_info* d_node_info, size_t node_infos_size) {
    // copy node_info from device to host
    hipMemcpy(node_info, d_node_info, node_infos_size, hipMemcpyDeviceToHost);
}

void augment_path_device(bool* do_change_capacity, Node_info* d_node_info, bool* d_do_change_capacity, u_int N, u_short* d_residual_capacity, u_int bottleneck_flow, size_t vertices_size) {
    int num_threads = 1024;
    int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    dim3 threads(num_threads);
    dim3 blocks(num_blocks);

    hipMemcpy(d_do_change_capacity, do_change_capacity, vertices_size, hipMemcpyHostToDevice);
    augment_path<<< blocks, threads >>>(d_node_info, d_do_change_capacity, N, d_residual_capacity, bottleneck_flow);
}

void free_device(u_short** d_residual_capacity, Node_info** d_node_info, bool** d_frontier, bool** d_visited) {
    hipFree(*d_residual_capacity);
    hipFree(*d_node_info);
    hipFree(*d_frontier);
    hipFree(*d_visited);
}