#include "hip/hip_runtime.h"
#include <bits/stdc++.h>
#include <stdio.h>
#include <stdlib.h>
#include "helpers.cuh"
using namespace std;

#define milliseconds 1e3
#define NUMPAD 128

typedef struct _Node_info{
    u_short parent_index;
    u_int potential_flow;
} Node_info;

u_int N;

void input(const char* filename, u_int total_nodes, u_short* residual_capacity);
void output(char* filename, int max_flow, double time);
__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited,
    u_int total_nodes, u_int sink, u_int* locks);


__global__ void reset(Node_info* node_info, bool* frontier, bool* visited, int source, int total_nodes, u_int* locks){
    int id = blockIdx.x * blockDim.x + threadIdx.x;
    if(id < total_nodes){
        frontier[id] = (id == source);
        visited[id] = false;
        node_info[id].potential_flow = UINT_MAX;
        locks[id] = 0;
    }
}

__global__ void augment_path(Node_info* node_infos, bool* do_change_capacity , u_int total_nodes, u_short* residual_capacity, u_int bottleneck_flow){
    int node_id = blockIdx.x * blockDim.x + threadIdx.x;
    if(node_id < total_nodes && do_change_capacity[node_id]){
        Node_info* current_node_info = node_infos + node_id;
        residual_capacity[current_node_info->parent_index * total_nodes + node_id] -= bottleneck_flow;
        residual_capacity[node_id * total_nodes + current_node_info->parent_index] += bottleneck_flow;
    }
}

void reset_host(bool* frontier, int source, int total_nodes, bool* do_change_capacity){
    frontier[source] = true;
    do_change_capacity[source] = false;

    for (int i = source+1; i < total_nodes; i++) {
        frontier[i] = false;
        do_change_capacity[i] = false;
    }

    for (int i = 0; i < source; i++) {
        frontier[i] = false;
        do_change_capacity[i] = false;
    }
}

__global__ void dev_is_frontier_empty_or_sink_found(bool* frontier, int i, int sink, int* found, int* cnt){
    if (!(*found)) {
        if (frontier[i]) {
            atomicExch(found, (i == sink));
            atomicAdd(cnt, 1);
        }
    }
    // if (frontier[i]) {
    //     atomicExch(found, (i == sink ? 1 : *found));
    //     atomicAdd(cnt, 1);
    // }
}

bool is_frontier_empty_or_sink_found(bool* frontier, int N, int sink_pos){
    for (int i = N-1; i > -1; --i) {
        if(frontier[i]){
            return i == sink_pos;
        }
    }
    return true;
}

int main(int argc, char** argv){
    if(argc < 4){
        printf("Specify filename & number of vertices\n");
        return 1;
    }

    Timer timer;

    N = atoi(argv[2]);
    // u_int V = atoi(argv[2]);
    // N = (V % NUMPAD == 0) ? V : (V / NUMPAD + 1) * NUMPAD;
    size_t matrix_size = N * N * sizeof(u_short);

    u_short *residual_capacity;
    residual_capacity = (u_short *)malloc(matrix_size);
    memset(residual_capacity, 0, matrix_size);

    input(argv[1], N, residual_capacity);

    u_int source = 0, sink = N - 1;
    u_int current_vertex, bottleneck_flow;
    u_int max_flow = 0;

    clock_t start_time = clock();

    size_t node_infos_size = N * sizeof(Node_info);
    size_t vertices_size = N * sizeof(bool);
    size_t locks_size = N * sizeof(u_int);

    Node_info* current_node_info;
    Node_info* node_info = (Node_info *)malloc(node_infos_size);
    bool* frontier = (bool *)malloc(vertices_size);
    bool* do_change_capacity = (bool *)malloc(vertices_size);

    u_short* d_residual_capacity;
    Node_info* d_node_info;
    bool* d_frontier, *d_visited, *d_do_change_capacity;
    u_int* d_locks;

    hipMalloc((void **)&d_residual_capacity, matrix_size);
    hipMalloc((void **)&d_node_info,node_infos_size);
    hipMalloc((void **)&d_frontier, vertices_size);
    hipMalloc((void **)&d_visited, vertices_size);
    hipMalloc((void **)&d_do_change_capacity, vertices_size);
    hipMalloc((void **)&d_locks, locks_size);
    // printf("d_residual_capacity_size: %d,\nd_locks_size: %d,\nd_node_info_size: %d,\nd_frontier: %d,\nd_visited: %d,\nd_do_change_capacity: %d\n",
    //         matrix_size, locks_size, node_infos_size, vertices_size, vertices_size, vertices_size);

    hipMemcpy(d_residual_capacity, residual_capacity, matrix_size, hipMemcpyHostToDevice);

    int found_augmenting_path = 0;
    int go_through_num = 0, last_go_through_num = 0;
    int* d_is_empty_or_found;
    int* d_go_through_num;
    hipMalloc(&d_is_empty_or_found, sizeof(int));
    hipMalloc(&d_go_through_num, sizeof(int));

    // int threads = 256;
    // int blocks = ceil(N * 1.0 /threads);
    int num_threads = 1024;
    int num_blocks = (N / 1024 <= 0) ? 1 : (N + 1023) / 1024;
    dim3 threads(num_threads);
    dim3 blocks(num_blocks);

    // timer.start();
    do {
        // reset visited, frontier, node_info, locks
        reset<<<blocks, threads>>>(d_node_info, d_frontier, d_visited, source, N, d_locks);
        reset_host(frontier, source, N, do_change_capacity);

        /*********************************/
        /**********      OLD       *******/
        /*********************************/
        // while(!is_frontier_empty_or_sink_found(frontier, N, sink)){
		// 		// Invoke kernel
		// 		find_augmenting_path<<< blocks, threads >>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks);

		// 		// Copy back frontier from device
		// 		hipMemcpy(frontier, d_frontier, vertices_size, hipMemcpyDeviceToHost);
        //         printf("%d\n", found_augmenting_path);
		// }

		// found_augmenting_path = frontier[sink];
        // printf("d, %d\n", found_augmenting_path);
		// if(!found_augmenting_path){
		// 	break;
		// }

        found_augmenting_path = go_through_num = 0;
        hipMemcpy(d_is_empty_or_found, &found_augmenting_path, sizeof(int), hipMemcpyHostToDevice);
        hipMemcpy(d_go_through_num, &go_through_num, sizeof(int), hipMemcpyHostToDevice);
        while(!found_augmenting_path) {
                // printf("d\n");

                // Invoke kernel
                find_augmenting_path<<<blocks, threads>>>(d_residual_capacity, d_node_info, d_frontier, d_visited, N, sink, d_locks);

                int i = N - 1;
                for (; i > -1; --i) {
                    dev_is_frontier_empty_or_sink_found<<<1, 1>>>(d_frontier, i, sink, d_is_empty_or_found, d_go_through_num);
                }
                hipMemcpy(&found_augmenting_path, d_is_empty_or_found, sizeof(int), hipMemcpyDeviceToHost);
                hipMemcpy(&go_through_num, d_go_through_num, sizeof(int), hipMemcpyDeviceToHost);
                // printf("%d, go through %d points\n", found_augmenting_path, go_through_num);
                if (!found_augmenting_path && ((u_int)go_through_num == 0 || go_through_num == last_go_through_num)) break;
                last_go_through_num = go_through_num;
        }
        // printf("d, %d\n", found_augmenting_path);
        if(!found_augmenting_path){
            break;
        }
        hipMemcpy(frontier, d_frontier, vertices_size, hipMemcpyDeviceToHost);

        // copy node_info from device to host
        hipMemcpy(node_info, d_node_info, node_infos_size, hipMemcpyDeviceToHost);

        bottleneck_flow = node_info[sink].potential_flow;
        max_flow += bottleneck_flow;

        for(current_vertex = sink; current_vertex != source; current_vertex = current_node_info->parent_index){
            current_node_info = node_info + current_vertex;
            do_change_capacity[current_vertex] = true;
        }

        hipMemcpy(d_do_change_capacity, do_change_capacity, vertices_size, hipMemcpyHostToDevice);

        augment_path<<< blocks, threads >>>(d_node_info, d_do_change_capacity, N, d_residual_capacity, bottleneck_flow);

    } while(found_augmenting_path);

    printf("\nN = %d", N);
    printf("\nmaxflow %d\n", max_flow);
    double time_taken = ((double)clock() - start_time)/CLOCKS_PER_SEC * milliseconds; // in milliseconds
    printf("%f ms for thread size- %d\n", time_taken, num_threads);
    output(argv[3], max_flow, time_taken);
    // timer.stop("compute total algorithm");
    // output(argv[3], max_flow, double(timer.time));

    free(residual_capacity);
    free(frontier);
    free(node_info);

    hipFree(d_is_empty_or_found);
    hipFree(d_residual_capacity);
    hipFree(d_node_info);
    hipFree(d_frontier);
    hipFree(d_visited);

    return 0;
}

void input(const char* filename, u_int total_nodes, u_short* residual_capacity) {
    ifstream file;
    file.open(filename);
    if (!file) {
        printf("Error reading file.\n");
        exit(1);
    }

    string line;
    u_int source, destination;
    u_short capacity;
    while (file) {
        getline(file, line);

        if (line.empty()) {
            continue;
        }
        std::stringstream linestream(line);
        linestream >> source >> destination >> capacity;
        residual_capacity[source * total_nodes + destination] = capacity;
    }
    file.close();
}

void output(char* filename, int max_flow, double time) {
    FILE* outfile = fopen(filename, "w");
    fprintf(outfile, "Max Flow: %d\n", max_flow);
    fprintf(outfile, "Time(ms): %.4f\n", time);
    fclose(outfile);
}

__global__ void find_augmenting_path(u_short* residual_capacity, Node_info* node_info, bool* frontier, bool* visited,
    u_int total_nodes, u_int sink, u_int* locks){

    int node_id = blockIdx.x * blockDim.x + threadIdx.x;

    if(!frontier[sink] && node_id < total_nodes && frontier[node_id]){

        frontier[node_id] = false;
        visited[node_id] = true;

        Node_info *neighbour;
        Node_info current_node_info = node_info[node_id];
        u_int capacity;

        for (u_int i = node_id; i < total_nodes; ++i){

            if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                continue;
            }

            frontier[i] = true;
            locks[i] = 0;

            neighbour = node_info + i;
            neighbour->parent_index = node_id;
            neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
        }


        for (u_int i = 0; i < node_id; ++i){

            if(frontier[i] || visited[i] || ((capacity = residual_capacity[node_id * total_nodes + i]) <= 0)){
                continue;
            }

            if(atomicCAS(locks+i, 0 , 1) == 1 || frontier[i]){
                continue;
            }

            frontier[i] = true;
            locks[i] = 0;

            neighbour = node_info + i;
            neighbour->parent_index = node_id;
            neighbour->potential_flow =  min(current_node_info.potential_flow, capacity);
        }
    }
}